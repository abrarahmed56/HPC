
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

void vector_dot_product(double* dot_product_ptr, const double* a, const double* b, long N){
  double sum = 0;
  #pragma omp parallel for schedule(static) reduction(+:sum)
  for (long i = 0; i < N; i++) sum += a[i] * b[i];
  *dot_product_ptr = sum;
}

void matrix_vector_product(double* prod_pts, const double* v, const double* M, long N) {
  for (long i = 0; i < N; i++) {
    double prod = 0;
    vector_dot_product(&prod, v, &(M[i*N]), N);
    *(prod_pts+i) = prod;
  }
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

#define BLOCK_SIZE 1024

// Warp divergence
__global__ void reduction_kernel0(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x %   2 == 0) smem[threadIdx.x] += smem[threadIdx.x + 1];
  __syncthreads();
  if (threadIdx.x %   4 == 0) smem[threadIdx.x] += smem[threadIdx.x + 2];
  __syncthreads();
  if (threadIdx.x %   8 == 0) smem[threadIdx.x] += smem[threadIdx.x + 4];
  __syncthreads();
  if (threadIdx.x %  16 == 0) smem[threadIdx.x] += smem[threadIdx.x + 8];
  __syncthreads();
  if (threadIdx.x %  32 == 0) smem[threadIdx.x] += smem[threadIdx.x + 16];
  __syncthreads();
  if (threadIdx.x %  64 == 0) smem[threadIdx.x] += smem[threadIdx.x + 32];
  __syncthreads();
  if (threadIdx.x % 128 == 0) smem[threadIdx.x] += smem[threadIdx.x + 64];
  __syncthreads();
  if (threadIdx.x % 256 == 0) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x % 512 == 0) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x == 0) sum[blockIdx.x] = smem[threadIdx.x] + smem[threadIdx.x + 512];
}

// Shared memory bank conflicts
__global__ void reduction_kernel1(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x *   2] += smem[threadIdx.x *   2 +   1];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x *   4] += smem[threadIdx.x *   4 +   2];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x *   8] += smem[threadIdx.x *   8 +   4];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x *  16] += smem[threadIdx.x *  16 +   8];
  __syncthreads();
  if (threadIdx.x <  32) smem[threadIdx.x *  32] += smem[threadIdx.x *  32 +  16];
  __syncwarp();
  if (threadIdx.x <  16) smem[threadIdx.x *  64] += smem[threadIdx.x *  64 +  32];
  __syncwarp();
  if (threadIdx.x <   8) smem[threadIdx.x * 128] += smem[threadIdx.x * 128 +  64];
  __syncwarp();
  if (threadIdx.x <   4) smem[threadIdx.x * 256] += smem[threadIdx.x * 256 + 128];
  __syncwarp();
  if (threadIdx.x <   2) smem[threadIdx.x * 512] += smem[threadIdx.x * 512 + 256];
  __syncwarp();
  if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[512];
}

__global__ void reduction_kernel2(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void vector_dot_product_kernel2(double* sum, const double* a, const double* b, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx] * b[idx];
  else smem[threadIdx.x] = 0;
  
  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void matrix_vector_product_kernel2(double* sum, const double* M, const double* v, long N){
  __shared__ double smem[BLOCK_SIZE];

  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N*N) smem[threadIdx.x] = M[idx] * v[idx%N];
  else smem[threadIdx.x] = 0;

  int partition = idx % N;
  //if partition is 0, then reduce everything in this block
  //if partition is > 0, then reduce everything to the left of this thread
  //and everything to the right


  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }

}

int main() {
  //long N = (1UL<<25);
  long N = (1UL<<15);

  double *M;
  hipHostMalloc((void**)&M, N * N * sizeof(double), hipHostMallocDefault);
  double *v;
  hipHostMalloc((void**)&v, N * sizeof(double), hipHostMallocDefault);
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N * N; i++) {
    M[i] = 1.0/(i+1);
  }
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    v[i] = 1.0/(i+1);
  }

  //double dot_product_ref, sums;
  double* prod_ref = (double*) (malloc(N * sizeof(double)));
  double* prod;
  double tt = omp_get_wtime();
  //reduction(&sum_ref, x, N);
  //vector_dot_product(&dot_product_ref, x1, x2, N);
  matrix_vector_product(prod_ref, v, M, N);
  printf("CPU Bandwidth = %f GB/s\n", 1*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double *M_d, *v_d, *y_d;
  hipMalloc(&M_d, N*N*sizeof(double));
  hipMalloc(&v_d, N*sizeof(double));
  //long N_work = 1;
  //for (long i = (N+BLOCK_SIZE-1)/(BLOCK_SIZE); i > 1; i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)) N_work += i;
  //cudaMalloc(&y_d, N_work*sizeof(double)); // extra memory buffer for reduction across thread-blocks
  hipMalloc(&y_d, N*sizeof(double));

  hipMemcpyAsync(v_d, v, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(M_d, M, N*N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  tt = omp_get_wtime();


  double* sum_d = y_d;
  //long Nb = (N*N+BLOCK_SIZE-1)/(BLOCK_SIZE);

  //vector_dot_product_kernel2<<<Nb,BLOCK_SIZE>>>(sum_d, v_d, M_d[i], N);
  matrix_vector_product_kernel2<<<N,BLOCK_SIZE>>>(sum_d, v_d, M_d, N);

  /*
  //vector_dot_product_kernel2<<<Nb,BLOCK_SIZE>>>(sum_d, x1_d, x2_d, N);
  while (Nb > 1) {
    long N = Nb;
    Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
    reduction_kernel2<<<Nb,BLOCK_SIZE>>>(sum_d + N, sum_d, N);
    sum_d += N;
  }
  */

  hipMemcpyAsync(prod, sum_d, N*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", 1*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double diff = 0;
  for (long i = 0; i < N; i++) {
    diff += fabs(prod[i] - prod_ref[i]);
  }
  printf("Error = %f\n", diff);

  hipFree(M_d);
  hipFree(v_d);
  hipFree(y_d);
  hipHostFree(M);
  hipHostFree(v);

  return 0;
}